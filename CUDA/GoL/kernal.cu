#include <stdio.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>



bool * cudaPtA;
bool * cudaPtB;


__global__
void kernal(bool * current, bool * next, int width, int height){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row < 0 || row >= height)
        return;
    if(col < 0 || col >= width)
        return;
    


    int neighbors = 0;

    for(int i = -1; i < 2; i++){
        int tempRow = 0;
        if(row+i < 0){
            tempRow = height-1;
        }
        else if(row+i >= height){
            tempRow = 0;
        }
        else{
            tempRow = row + i;
        }

        for(int j = -1; j < 2; j++){
            int tempCol = 0;
            if(col+i < 0){
                tempCol = width-1;
            }
            else if(col+i >= width){
                tempCol = 0;
            }
            else{
                tempCol = col + j;
            }

            if(!(i == 0 && j == 0)){
                if(current[tempRow*width + tempCol]){
                    neighbors++;
                }
            }
        }
    }

    bool cell = current[row*width+col];

    if(!cell && neighbors == 3){
        next[row*width+col] = true;
    }
    else if(cell && (neighbors < 2 || neighbors > 3)){
        next[row*width+col] = false;
    }
    else if(cell && (neighbors == 2 || neighbors == 3)){
        next[row*width+col] = true;
    }       
}

extern void initCUDA(int size, bool * data){
    hipMalloc(&cudaPtA, size * sizeof(bool));
    hipMalloc(&cudaPtB, size * sizeof(bool));
}

extern void iteration(bool * data, int width, int height){
    int size = width*height;
    dim3 block(32,32);
    dim3 grid((width/32)+1, (height/32)+1);

    hipMemcpy(cudaPtA, data, size * sizeof(bool), hipMemcpyHostToDevice);
    kernal<<<grid, block>>>(cudaPtA, cudaPtB, width, height);
    hipMemcpy(data, cudaPtB, size * sizeof(bool), hipMemcpyDeviceToHost);
}

extern void exitCUDA(){
    hipFree(cudaPtA);
    hipFree(cudaPtB);
}