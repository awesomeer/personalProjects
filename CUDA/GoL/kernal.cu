#include <stdio.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


//static unsigned char turn;
static unsigned char * cudaPtA;
static unsigned char * cudaPtB;


__global__
void kernal(unsigned char * current, unsigned char * next, int width, int height){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row < 0 || row >= height)
        return;
    if(col < 0 || col >= width)
        return;
    


    int neighbors = 0;

    for(int i = -1; i < 2; i++){
        int tempRow = 0;
        if(row+i < 0){
            tempRow = height-1;
        }
        else if(row+i >= height){
            tempRow = 0;
        }
        else{
            tempRow = row + i;
        }

        for(int j = -1; j < 2; j++){
            int tempCol = 0;
            if(col+i < 0){
                tempCol = width-1;
            }
            else if(col+i >= width){
                tempCol = 0;
            }
            else{
                tempCol = col + j;
            }

            if(!(i == 0 && j == 0)){
                if(current[tempRow*width + tempCol] == 255){
                    neighbors++;
                }
            }
        }
    }

    int cell = current[row*width+col];

    if(cell == 0 && neighbors == 3){
        next[row*width+col] = 255;
    }
    else if(cell == 255 && (neighbors < 2 || neighbors > 3)){
        next[row*width+col] = 0;
    }
    else if(cell == 255 && (neighbors == 2 || neighbors == 3)){
        next[row*width+col] = 255;
    }       
}

extern void initCUDA(int size, unsigned char * data){
    //turn = 0;
    hipMalloc(&cudaPtA, size * sizeof(unsigned char));
    hipMalloc(&cudaPtB, size * sizeof(unsigned char));
    hipMemcpy(cudaPtA, data, size * sizeof(unsigned char), hipMemcpyHostToDevice);
}

extern void iteration(unsigned char * data, int width, int height){
    int size = width*height;
    dim3 block(32,32);
    dim3 grid((width/32)+1, (height/32)+1);

    hipMemcpy(cudaPtA, data, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    kernal<<<grid, block>>>(cudaPtA, cudaPtB, width, height);
    hipMemcpy(data, cudaPtB, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
}

extern void exitCUDA(){
    hipFree(cudaPtA);
    hipFree(cudaPtB);
}