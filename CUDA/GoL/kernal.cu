#include <stdlib.h>

#include <hip/hip_runtime.h>



static bool * cudaPtA;
static bool * cudaPtB;
static unsigned char * renderImage;


__global__
void kernal(bool * current, bool * next, int width, int height){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row < 0 || row >= height)
        return;
    if(col < 0 || col >= width)
        return;
    


    int neighbors = 0;

    for(int i = -1; i < 2; i++){
        int tempRow = 0;
        if(row+i < 0){
            tempRow = height-1;
        }
        else if(row+i >= height){
            tempRow = 0;
        }
        else{
            tempRow = row + i;
        }

        for(int j = -1; j < 2; j++){
            int tempCol = 0;
            if(col+i < 0){
                tempCol = width-1;
            }
            else if(col+i >= width){
                tempCol = 0;
            }
            else{
                tempCol = col + j;
            }

            if(!(i == 0 && j == 0)){
                if(current[tempRow*width + tempCol]){
                    neighbors++;
                }
            }
        }
    }

    bool cell = current[row*width+col];

    if(!cell && neighbors == 3){
        next[row*width+col] = true;
    }
    else if(cell && (neighbors < 2 || neighbors > 3)){
        next[row*width+col] = false;
    }
    else if(cell && (neighbors == 2 || neighbors == 3)){
        next[row*width+col] = true;
    }

}


__global__
void render(bool* state, unsigned char* image, int width, int height) {

    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < 0 || row >= height)
        return;
    if (col < 0 || col >= width)
        return;


    int step = width * height;
    int index = row * width + col;

    if (state[index]){
        image[index] = row * 256/height;
        image[index + step] = col*256/width;
        image[index + (2 * step)] = 255;
    }
    else{
        image[index] = 0;
        image[index + step] = 0;
        image[index + (2 * step)] = 0;
    }
}



extern void randomize(int size) {
    bool* life = (bool*)malloc(sizeof(bool) * size);
    for (int i = 0; i < size; i++) {
        life[i] = rand() / (RAND_MAX / 2);
    }

    hipMemcpy(cudaPtA, life, sizeof(bool) * size, hipMemcpyHostToDevice);

    free(life);
}

extern void initCUDA(int size){

    hipMalloc(&cudaPtA, size * sizeof(bool));
    hipMalloc(&cudaPtB, size * sizeof(bool));
    hipMalloc(&renderImage, 3 * size * sizeof(unsigned char));

    randomize(size);
}



extern void iteration(unsigned char * image, int width, int height){
    int size = width*height;
    dim3 block(32,32);
    dim3 grid((width/32)+1, (height/32)+1);
        
    //Iterate one step in simulation
    kernal<<<grid, block>>>(cudaPtA, cudaPtB, width, height);
    hipMemcpy(cudaPtA, cudaPtB, size * sizeof(bool), hipMemcpyDeviceToDevice);

    //Render image
    render<<<grid, block>>>(cudaPtB, renderImage, width, height);
    hipMemcpy(image, renderImage, 3 * size * sizeof(unsigned char), hipMemcpyDeviceToHost);

}

extern void exitCUDA(){
    hipFree(cudaPtA);
    hipFree(cudaPtB);
    hipFree(renderImage);
}