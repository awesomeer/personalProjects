#include <stdlib.h>

#include <hip/hip_runtime.h>


#include <math.h>


__managed__ int cuWidth, cuHeight;
static bool * cuCurr;
static bool * cuNext;
static unsigned char * renderImage;

__device__
int wrap(int index, int size);

__device__
int hashSubImage(int row, int col, int xLen, int yLen);


__global__
void kernal(bool * current, bool * next, unsigned char * image){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row < 0 || row >= cuHeight)
        return;
    if(col < 0 || col >= cuWidth)
        return;
    


    int neighbors = 0;

    for(int i = -1; i < 2; i++){
        int tempRow = wrap(row + i, cuHeight);

        for(int j = -1; j < 2; j++){
            int tempCol = wrap(col + j, cuWidth);

            if(!(i == 0 && j == 0)){
                if(current[tempRow* cuWidth + tempCol]){
                    neighbors++;
                }
            }
        }
    }

    int index = row * cuWidth + col;
    bool cell = current[index];

    unsigned char* red = image;
    unsigned char* green = &image[cuWidth * cuHeight];
    unsigned char* blue = &image[2 * cuWidth * cuHeight];

    //if(!cell && neighbors == 3){
    //    next[index] = true;
    //}
    //else if(cell && (neighbors < 2 || neighbors > 3)){
    //    next[index] = false;
    //}
    //else if(cell && (neighbors == 2 || neighbors == 3)){
    //    next[index] = true;
    //}

    if ((!cell && neighbors == 3) || (cell && (neighbors == 2 || neighbors == 3))) {
        next[index] = true;
        red[index] = green[index] = blue[index] = 255;
    }
    else if (cell && (neighbors < 2 || neighbors > 3)) {
        next[index] = false;
        red[index] = green[index] = blue[index] = 0;
    }

}


__global__
void render(bool* state, unsigned char* image) {

    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < 0 || row >= cuHeight)
        return;
    if (col < 0 || col >= cuWidth)
        return;


    int step = cuWidth * cuHeight;
    unsigned char* red = image;
    unsigned char* green = &image[step];
    unsigned char* blue = &image[2 * step];

    int index = row * cuWidth + col;

    if (state[index]) {
        red[index] = 256 * row / cuHeight;
        green[index] = 256 * col / cuWidth;
        blue[index] = 255;
    }

}


__device__
int hashSubImage(unsigned char * image, int row, int col, int xLen, int yLen) {
    int hashSum = 0;

    for (int y = 0; y < yLen; y++) {
        int yHash = 0;
        if (y < ceil(yLen / 2.0))
            yHash = y;
        else
            yHash = yLen - y - 1;

        for (int x = 0; x < xLen; x++) {
            int xHash = 0;
            if (x < ceil(xLen / 2.0))
                xHash = x;
            else
                xHash = xLen - x - 1;

            int hash = yHash > xHash ? xHash : yHash;
        }
    }
}


__device__
int wrap(int index, int size) {

    while (index < 0)
        index += size;

    return index % size;
}


extern void randomize(int size) {
    bool* life = (bool*)malloc(sizeof(bool) * size);
    for (int i = 0; i < size; i++) {
        life[i] = rand() / (RAND_MAX / 2);
    }

    hipMemcpy(cuCurr, life, sizeof(bool) * size, hipMemcpyHostToDevice);

    free(life);
}

extern void initCUDA(int width, int height){
    int size = width * height;
    cuWidth = width;
    cuHeight = height;

    hipMalloc(&cuCurr, size * sizeof(bool));
    hipMalloc(&cuNext, size * sizeof(bool));
    hipMalloc(&renderImage, 3 * size * sizeof(unsigned char));

    randomize(size);
}


extern void iteration(unsigned char * image){
    int size = cuWidth * cuHeight;
    dim3 block(32,32);
    dim3 grid((cuWidth/32)+1, (cuHeight/32)+1);
        
    //Iterate one step in simulation
    kernal<<<grid, block>>>(cuCurr, cuNext, renderImage);
    hipMemcpy(cuCurr, cuNext, size * sizeof(bool), hipMemcpyDeviceToDevice);

    //Render image
    render<<<grid, block>>>(cuNext, renderImage);
    hipMemcpy(image, renderImage, 3 * size * sizeof(unsigned char), hipMemcpyDeviceToHost);

}

extern void exitCUDA(){
    hipFree(cuCurr);
    hipFree(cuNext);
    hipFree(renderImage);
}