#include <stdlib.h>

#include <hip/hip_runtime.h>



static bool * cudaPtA;
static bool * cudaPtB;
static unsigned char * renderImage;

static bool * life;


__global__
void kernal(bool * current, bool * next, int width, int height){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row < 0 || row >= height)
        return;
    if(col < 0 || col >= width)
        return;
    


    int neighbors = 0;

    for(int i = -1; i < 2; i++){
        int tempRow = 0;
        if(row+i < 0){
            tempRow = height-1;
        }
        else if(row+i >= height){
            tempRow = 0;
        }
        else{
            tempRow = row + i;
        }

        for(int j = -1; j < 2; j++){
            int tempCol = 0;
            if(col+i < 0){
                tempCol = width-1;
            }
            else if(col+i >= width){
                tempCol = 0;
            }
            else{
                tempCol = col + j;
            }

            if(!(i == 0 && j == 0)){
                if(current[tempRow*width + tempCol]){
                    neighbors++;
                }
            }
        }
    }

    bool cell = current[row*width+col];

    if(!cell && neighbors == 3){
        next[row*width+col] = true;
    }
    else if(cell && (neighbors < 2 || neighbors > 3)){
        next[row*width+col] = false;
    }
    else if(cell && (neighbors == 2 || neighbors == 3)){
        next[row*width+col] = true;
    }       
}


__global__
void render(bool* state, unsigned char* image, int width, int height) {
    //int row = threadIdx.y + blockIdx.y * blockDim.y;
    //int col = threadIdx.x + blockIdx.x * blockDim.x;

    int step = width * height;
    //int index = row * width + col;

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= width * height)
        return;

    if (state[index]){
        image[index] = 255;
        image[index + step] = 255;
        image[index + (2 * step)] = 255;
    }
    else{
        image[index] = 0;
        image[index + step] = 0;
        image[index + (2 * step)] = 0;
    }
}


extern void randomize(int size) {
    for (int i = 0; i < size; i++) {
        life[i] = rand() > (RAND_MAX / 2);
    }
}

extern void initCUDA(int size){

    life = (bool *) malloc(sizeof(bool) * size);
    randomize(size);

    hipMalloc(&cudaPtA, size * sizeof(bool));
    hipMalloc(&cudaPtB, size * sizeof(bool));
    hipMalloc(&renderImage, 3 * size * sizeof(unsigned char));
}



extern void iteration(unsigned char * image, int width, int height){
    int size = width*height;
    dim3 block(32,32);
    dim3 grid((width/32)+1, (height/32)+1);
        
    //Iterate one step in simulation
    hipMemcpy(cudaPtA, life, size * sizeof(bool), hipMemcpyHostToDevice);
    kernal<<<grid, block>>>(cudaPtA, cudaPtB, width, height);
    hipMemcpy(life, cudaPtB, size * sizeof(bool), hipMemcpyDeviceToHost);

    //Render image
    render<<<(width*height/1024)+1, 1024>>>(cudaPtB, renderImage, width, height);
    hipMemcpy(image, renderImage, 3 * size * sizeof(unsigned char), hipMemcpyDeviceToHost);
}

extern void exitCUDA(){
    free(life);
    hipFree(cudaPtA);
    hipFree(cudaPtB);
    hipFree(renderImage);
}