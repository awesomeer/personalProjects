#include <stdlib.h>

#include <hip/hip_runtime.h>


#include <math.h>

static bool * cudaPtA;
static bool * cudaPtB;
static unsigned char * renderImage;


__global__
void kernal(bool * current, bool * next, int width, int height){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row < 0 || row >= height)
        return;
    if(col < 0 || col >= width)
        return;
    


    int neighbors = 0;

    for(int i = -1; i < 2; i++){
        int tempRow = 0;
        if(row+i < 0){
            tempRow = height-1;
        }
        else if(row+i >= height){
            tempRow = 0;
        }
        else{
            tempRow = row + i;
        }

        for(int j = -1; j < 2; j++){
            int tempCol = 0;
            if(col+i < 0){
                tempCol = width-1;
            }
            else if(col+i >= width){
                tempCol = 0;
            }
            else{
                tempCol = col + j;
            }

            if(!(i == 0 && j == 0)){
                if(current[tempRow*width + tempCol]){
                    neighbors++;
                }
            }
        }
    }

    bool cell = current[row*width+col];

    if(!cell && neighbors == 3){
        next[row*width+col] = true;
    }
    else if(cell && (neighbors < 2 || neighbors > 3)){
        next[row*width+col] = false;
    }
    else if(cell && (neighbors == 2 || neighbors == 3)){
        next[row*width+col] = true;
    }

}


__global__
void render(bool* state, unsigned char* image, int width, int height) {

    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < 0 || row >= height)
        return;
    if (col < 0 || col >= width)
        return;


    int step = width * height;
    unsigned char* red = image;
    unsigned char* green = &image[step];
    unsigned char* blue = &image[2 * step];

    int index = row * width + col;

    if (state[index]){

        /* Weird Circular render
        red[index] = 256 * sin(3.1415 * row / height);//row * 256 / height;
        green[index] = 256 * sin(3.1415 * col / width);//col * 256 / width;
        blue[index] = 256 * sin(3.1415 * (row * width) / (height*width));
        */

         //Straight rectangular render
        red[index] = row * 256 / height;
        green[index] = col * 256 / width;
        blue[index] = 255;
        
    }
    else{
        red[index] = 0;
        green[index] = 0;
        blue[index] = 0;
    }
}



extern void randomize(int size) {
    bool* life = (bool*)malloc(sizeof(bool) * size);
    for (int i = 0; i < size; i++) {
        life[i] = rand() / (RAND_MAX / 2);
    }

    hipMemcpy(cudaPtA, life, sizeof(bool) * size, hipMemcpyHostToDevice);

    free(life);
}

extern void initCUDA(int size){

    hipMalloc(&cudaPtA, size * sizeof(bool));
    hipMalloc(&cudaPtB, size * sizeof(bool));
    hipMalloc(&renderImage, 3 * size * sizeof(unsigned char));

    randomize(size);
}



extern void iteration(unsigned char * image, int width, int height){
    int size = width*height;
    dim3 block(32,32);
    dim3 grid((width/32)+1, (height/32)+1);
        
    //Iterate one step in simulation
    kernal<<<grid, block>>>(cudaPtA, cudaPtB, width, height);
    hipMemcpy(cudaPtA, cudaPtB, size * sizeof(bool), hipMemcpyDeviceToDevice);

    //Render image
    render<<<grid, block>>>(cudaPtB, renderImage, width, height);
    hipMemcpy(image, renderImage, 3 * size * sizeof(unsigned char), hipMemcpyDeviceToHost);

}

extern void exitCUDA(){
    hipFree(cudaPtA);
    hipFree(cudaPtB);
    hipFree(renderImage);
}