#include <stdlib.h>

#include <hip/hip_runtime.h>


#include <math.h>

static bool * cudaPtA;
static bool * cudaPtB;
static unsigned char * renderImage;


__global__
void kernal(bool * current, bool * next, int width, int height, unsigned char * image){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row < 0 || row >= height)
        return;
    if(col < 0 || col >= width)
        return;
    


    int neighbors = 0;

    for(int i = -1; i < 2; i++){
        int tempRow = 0;
        if(row+i < 0){
            tempRow = height-1;
        }
        else if(row+i >= height){
            tempRow = 0;
        }
        else{
            tempRow = row + i;
        }

        for(int j = -1; j < 2; j++){
            int tempCol = 0;
            if(col+j < 0){
                tempCol = width-1;
            }
            else if(col+j >= width){
                tempCol = 0;
            }
            else{
                tempCol = col + j;
            }

            if(!(i == 0 && j == 0)){
                if(current[tempRow*width + tempCol]){
                    neighbors++;
                }
            }
        }
    }

    int index = row * width + col;
    bool cell = current[index];

    if(!cell && neighbors == 3){
        next[index] = true;
    }
    else if(cell && (neighbors < 2 || neighbors > 3)){
        next[index] = false;
    }
    else if(cell && (neighbors == 2 || neighbors == 3)){
        next[index] = true;
    }
        
    int step = width * height;
    unsigned char* red = image;
    unsigned char* green = &image[step];
    unsigned char* blue = &image[2 * step];

    if (next[index]) {
        red[index] = 255;
        green[index] = 255;
        blue[index] = 255;
    }
}


__global__
void render(bool* state, unsigned char* image, int width, int height) {

    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < 0 || row >= height)
        return;
    if (col < 0 || col >= width)
        return;


    int step = width * height;
    unsigned char* red = image;
    unsigned char* green = &image[step];
    unsigned char* blue = &image[2 * step];

    int index = row * width + col;

    if (state[index]){
         //Straight rectangular render
        red[index] = row * 256 / height;
        green[index] = col * 256 / width;
        //blue[index] = 255;
    }
}



extern void randomize(int size) {
    bool* life = (bool*)malloc(sizeof(bool) * size);
    for (int i = 0; i < size; i++) {
        life[i] = rand() / (RAND_MAX / 2);
    }

    hipMemcpy(cudaPtA, life, sizeof(bool) * size, hipMemcpyHostToDevice);

    free(life);
}

extern void initCUDA(int size){

    hipMalloc(&cudaPtA, size * sizeof(bool));
    hipMalloc(&cudaPtB, size * sizeof(bool));
    hipMalloc(&renderImage, 3 * size * sizeof(unsigned char));

    randomize(size);
}



extern void iteration(unsigned char * image, int width, int height){
    int size = width*height;
    dim3 block(32,32);
    dim3 grid((width/32)+1, (height/32)+1);
        
    //Iterate one step in simulation
    hipMemset(renderImage, 0, 3 * size * sizeof(unsigned char));
    kernal<<<grid, block>>>(cudaPtA, cudaPtB, width, height, renderImage);
    hipMemcpy(cudaPtA, cudaPtB, size * sizeof(bool), hipMemcpyDeviceToDevice);

    //Render image
    render<<<grid, block>>>(cudaPtB, renderImage, width, height);
    hipMemcpy(image, renderImage, 3 * size * sizeof(unsigned char), hipMemcpyDeviceToHost);

}

extern void exitCUDA(){
    hipFree(cudaPtA);
    hipFree(cudaPtB);
    hipFree(renderImage);
}