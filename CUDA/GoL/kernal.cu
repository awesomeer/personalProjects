#include <stdlib.h>

#include <hip/hip_runtime.h>


#include <math.h>


__managed__ int cuWidth, cuHeight;
static bool * cudaPtA;
static bool * cudaPtB;
static unsigned char * renderImage;


__global__
void kernal(bool * current, bool * next){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row < 0 || row >= cuHeight)
        return;
    if(col < 0 || col >= cuWidth)
        return;
    


    int neighbors = 0;

    for(int i = -1; i < 2; i++){
        int tempRow = 0;
        if(row+i < 0){
            tempRow = cuHeight-1;
        }
        else if(row+i >= cuHeight){
            tempRow = 0;
        }
        else{
            tempRow = row + i;
        }

        for(int j = -1; j < 2; j++){
            int tempCol = 0;
            if(col+j < 0){
                tempCol = cuWidth-1;
            }
            else if(col+j >= cuWidth){
                tempCol = 0;
            }
            else{
                tempCol = col + j;
            }

            if(!(i == 0 && j == 0)){
                if(current[tempRow* cuWidth + tempCol]){
                    neighbors++;
                }
            }
        }
    }

    int index = row * cuWidth + col;
    bool cell = current[index];

    if(!cell && neighbors == 3){
        next[index] = true;
    }
    else if(cell && (neighbors < 2 || neighbors > 3)){
        next[index] = false;
    }
    else if(cell && (neighbors == 2 || neighbors == 3)){
        next[index] = true;
    }
        
}


__global__
void render(bool* state, unsigned char* image) {

    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < 0 || row >= cuHeight)
        return;
    if (col < 0 || col >= cuWidth)
        return;


    int step = cuWidth * cuHeight;
    unsigned char* red = image;
    unsigned char* green = &image[step];
    unsigned char* blue = &image[2 * step];

    int index = row * cuWidth + col;

    if (state[index]) {
        red[index] = 256 * row / cuHeight;
        green[index] = 256 * col / cuWidth;
        blue[index] = 255;
    }
    else {
        red[index] = 0;
        green[index] = 0;
        blue[index] = 0;
    }
}



extern void randomize(int size) {
    bool* life = (bool*)malloc(sizeof(bool) * size);
    for (int i = 0; i < size; i++) {
        life[i] = rand() / (RAND_MAX / 2);
    }

    hipMemcpy(cudaPtA, life, sizeof(bool) * size, hipMemcpyHostToDevice);

    free(life);
}

extern void initCUDA(int width, int height){
    int size = width * height;
    cuWidth = width;
    cuHeight = height;

    hipMalloc(&cudaPtA, size * sizeof(bool));
    hipMalloc(&cudaPtB, size * sizeof(bool));
    hipMalloc(&renderImage, 3 * size * sizeof(unsigned char));

    randomize(size);
}



extern void iteration(unsigned char * image){
    int size = cuWidth * cuHeight;
    dim3 block(32,32);
    dim3 grid((cuWidth/32)+1, (cuHeight/32)+1);
        
    //Iterate one step in simulation
    kernal<<<grid, block>>>(cudaPtA, cudaPtB);
    hipMemcpy(cudaPtA, cudaPtB, size * sizeof(bool), hipMemcpyDeviceToDevice);

    //Render image
    render<<<grid, block>>>(cudaPtB, renderImage);
    hipMemcpy(image, renderImage, 3 * size * sizeof(unsigned char), hipMemcpyDeviceToHost);

}

extern void exitCUDA(){
    hipFree(cudaPtA);
    hipFree(cudaPtB);
    hipFree(renderImage);
}