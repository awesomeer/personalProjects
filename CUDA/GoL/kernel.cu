#include "hip/hip_runtime.h"
#include "kernel.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>


__managed__ static int width;
__managed__ static int height;


#define SIZE (width*height)

bool* current, *next;
unsigned char* frame;



__global__
void randomize(bool* curr) {
	int index = threadIdx.x + 1024 * blockIdx.x;
	curr[index] = rand() % 2;
}


//Wrapper Function to call the GPU kernel
void randomize(void) {
	randomize << <SIZE/1024, 1024 >> > (current);
}

void initCUDA(int swidth, int sheight, unsigned int seed) {
	width = swidth;
	height = sheight;

	srand(seed);
	hipMalloc(&current, sizeof(bool) * width * height); //Allocate memory for current world
	hipMalloc(&next, sizeof(bool) * width * height); //Allocate memory to store next iteration
	hipMalloc(&frame, sizeof(unsigned char) * width * height); //Memory to store the rendered frame

	randomize();
}

void iterate(unsigned char* world) {

}

void freeCUDA(void);