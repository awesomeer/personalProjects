#include "hip/hip_runtime.h"
#include "kernel.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <stdio.h>
#include <math.h>

__managed__ static int width;
__managed__ static int height;


#define SIZE (width*height)

bool* current, *next;
unsigned char* frame;


void randomize(void) {
	bool* temp = (bool *) malloc(sizeof(bool) * SIZE);
	for (int i = 0; i < SIZE; i++)
		temp[i] = rand() % 2;

	//temp[0] = true;
	//temp[1] = true;
	//temp[width] = true;

	//temp[3 + width * 2] = true;
	//temp[3 + width * 3] = true;
	//temp[2 + width * 3] = true;

	hipMemcpy(current, temp, SIZE * sizeof(bool), hipMemcpyHostToDevice);
	free(temp);
}

void initCUDA(int swidth, int sheight) {
	width = swidth;
	height = sheight;

	hipMalloc(&current, sizeof(bool) * SIZE); //Allocate memory for current world
	hipMalloc(&next, sizeof(bool) * SIZE); //Allocate memory to store next iteration
	hipMalloc(&frame, 3 * sizeof(unsigned char) * SIZE); //Memory to store the rendered frame

	randomize();
}

__device__ __forceinline__
int wrap(int val, int limit) {
	if (val < 0)
		return limit - 1;
	return val % limit;
}

__device__ __forceinline__
int index(int x, int y) {
	return wrap(x, width) + width * wrap(y, height);
}

__device__ __forceinline__
int red(int x, int y) {
	return index(x, y);
}

__device__ __forceinline__
int green(int x, int y) {
	return index(x, y) + SIZE;
}

__device__ __forceinline__
int blue(int x, int y) {
	return index(x, y) + 2 * SIZE;
}


__global__
void computeIteration(bool* current, bool*next, unsigned char* frame) {

	int x = threadIdx.x + 32 * blockIdx.x;
	int y = threadIdx.y + 32 * blockIdx.y;

	if (x >= width || y >= height)
		return;

	//Compute the neighbors each cell has
	char neighbors = 0;
	for (int r = -1; r < 2; r++) {
		int tempY = wrap(y + r, height);

		for (int c = -1; c < 2; c++) {
			int tempX = wrap(x + c, width);

			if (current[index(tempX, tempY)] && !(r == 0 && c == 0))
				neighbors++;
		}
	}


	//Check the rules for cell death and birth
	if (current[index(x, y)] && (neighbors == 2 || neighbors == 3)) {
		next[index(x, y)] = true;
		frame[red(x, y)] = 255;
		frame[green(x, y)] = 255;
		frame[blue(x, y)] = 255;
	}
	else if (!current[index(x, y)] && neighbors == 3) {
		next[index(x, y)] = true;
		frame[red(x, y)] = 255;
		frame[green(x, y)] = 255;
		frame[blue(x, y)] = 255;
	}
	else {
		next[index(x, y)] = false;
		frame[red(x, y)] = 0;
		frame[green(x, y)] = 0;
		frame[blue(x, y)] = 0;
	}
}


__device__
void colorBlock(unsigned char* frame, int x, int y) {

	for (int i = 0; i < 2; i++) {
		for (int j = 0; j < 2; j++) {
			frame[red(x + j, y + i)] = 255;
			frame[green(x + j, y + i)] = 0;
			frame[blue(x + j, y + i)] = 0;
		}
	}
}

__device__
void color5x5(unsigned char* frame, int x, int y, unsigned char redC, unsigned char greenC, unsigned char blueC) {

	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			if (frame[red(x + j, y + i)] == 255 && frame[green(x + j, y + i)] == 255 && frame[blue(x + j, y + i)] == 255) {
				frame[red(x + j, y + i)] = redC;
				frame[green(x + j, y + i)] = greenC;
				frame[blue(x + j, y + i)] = blueC;
			}
		}
	}
}

__device__
void color6x6(unsigned char* frame, int x, int y, unsigned char redC, unsigned char greenC, unsigned char blueC) {

	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			if (frame[red(x + j, y + i)] == 255 && frame[green(x + j, y + i)] == 255 && frame[blue(x + j, y + i)] == 255) {
				frame[red(x + j, y + i)] = redC;
				frame[green(x + j, y + i)] = greenC;
				frame[blue(x + j, y + i)] = blueC;
			}
		}
	}
}


__device__
int computeHash(bool* world, int x, int y, int hashWidth, int hashHeight) {
	int acc = 0;

	for (int i = 0; i < hashHeight; i++) {
		int tempI = i;
		if (i > (hashHeight - 1) / 2)
			tempI = hashHeight - i - 1;
		for (int j = 0; j < hashWidth; j++) {
			int tempJ = j;
			if (j > (hashWidth - 1) / 2)
				tempJ = hashWidth - j - 1;

			if (world[index(x + j, y + i)]) {
				int add = tempI > tempJ ? tempJ : tempI;
				if (i == 0 || j == 0 || i == hashHeight - 1 || j == hashWidth - 1)
					return 0;
				acc += add * add * add;
			}
		}
	}
	
	return acc;
}


__global__
void renderFancy(bool* world, unsigned char* frame) {

	int x = threadIdx.x + 32 * blockIdx.x;
	int y = threadIdx.y + 32 * blockIdx.y;

	if (x >= width || y >= height)
		return;


	int hash = 0;
	hash = computeHash(world, x, y, 6, 6);
	switch (hash) {
	case 14: color6x6(frame, x + 1, y + 1, 255, 0, 255); break;//Loaf Red + Blue = Magenta
	case 22:
	case 34:
	case 6: {
		color6x6(frame, x + 1, y + 1, 0, 255, 255); break;//Toad and Beacon Green + Blue = Cyan
	}
	}

	__syncthreads();

	hash = computeHash(world, x, y, 5, 5);
	switch (hash) {
	case 4: color5x5(frame, x + 1, y + 1, 0, 255, 0); break;//Tub Green
	case 5: color5x5(frame, x + 1, y + 1, 255, 255, 0); break; //Boat Red+Green = Yellow
	case 10: color5x5(frame, x + 1, y + 1, 0, 0, 255); break; //Blinker Blue
	}

	__syncthreads();

	hash = computeHash(world, x, y, 4, 4);
	if (hash == 4)
		colorBlock(frame, x + 1, y + 1);

}

void iteration(unsigned char* world) {

	dim3 block = dim3((width / 32) + 1, (height / 32) + 1);
	dim3 thread = dim3(32, 32);

	computeIteration<<<block, thread>>>(current, next, frame);
	renderFancy<<<block, thread>>>(next, frame);
	hipDeviceSynchronize();

	hipMemcpy(world, frame, 3 * sizeof(unsigned char) * SIZE, hipMemcpyDeviceToHost);
	bool* temp = current;
	current = next;
	next = temp;
	
}

void freeCUDA(void) {
	hipFree(current);
	hipFree(next);
	hipFree(frame);
}


