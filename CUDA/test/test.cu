#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define SIZE 10

__global__
void inc(int * array){
	int index = threadIdx.x + (blockDim.x * blockIdx.x);
	array[index]++;
}


int main(){

	int input[SIZE];
	for(int i = 0; i < SIZE; i++)
		input[i] = i * 3;
	
	int * cudaInput;
	hipMalloc(&cudaInput, SIZE * sizeof(int));
	hipMemcpy(cudaInput, input, SIZE * sizeof(int), hipMemcpyHostToDevice);

	inc<<<1, 10>>>(cudaInput);

	hipError_t err = hipGetLastError();
	printf("%s\n", hipGetErrorString(err));
	hipMemcpy(input, cudaInput, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 0 ; i < SIZE; i++){
		printf("%d: %d\n", i, input[i]);
	}

}

