#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__
void inc(int * array){
	int index = threadIdx.x + (blockDim.x * blockIdx.x);
	array[index]++;
}


int main(){

	int * cudaInput;
	hipMallocManaged(&cudaInput, sizeof(int) * 20);
	for(int i = 0; i < 20; i++){
		cudaInput[i] = i * 2;
	}

	for(int i = 0; i < 20; i++){
		printf("%d: %d\n", i, cudaInput[i]);
	}

	inc<<<1, 20>>>(cudaInput);
	inc<<<1, 20>>>(cudaInput);

	hipError_t error = hipGetLastError();
	printf("%s\n", hipGetErrorString(error));
	
	for(int i = 0; i < 20; i++){
		printf("%d: %d\n", i, cudaInput[i]);
	}
	
	hipFree(cudaInput);

}

