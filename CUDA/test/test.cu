#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__
void inc(int * array){
	int index = threadIdx.x + (blockDim.x * blockIdx.x);
	array[index]++;
}


int main(){
	int input[20];
	for(int i = 0; i < 20; i++){
		input[i] = i*2;
	}

	int * cudaInput;
	hipMalloc(&cudaInput, sizeof(int) * 20);
	hipMemcpy(cudaInput, input, sizeof(int) * 20, hipMemcpyHostToDevice);
	inc<<<1, 20>>>(cudaInput);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	printf("%s\n", hipGetErrorString(error));
	hipMemcpy(input, cudaInput, sizeof(int) * 20, hipMemcpyDeviceToHost);
	
	for(int i = 0; i < 20; i++){
		printf("%d: %d\n", i, input[i]);
	}
	
	hipFree(cudaInput);

}

