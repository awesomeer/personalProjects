#include "hip/hip_runtime.h"
#include "../inc/kernel.h"

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include <stdio.h>


/*
* This code is configured for a GTX 1060M with 1024 CUDA cores
*/


#define HEIGHT 720
#define WIDTH 1280
#define SIZE (3 * WIDTH * HEIGHT)


__managed__ char GX[9] = { 1, 0, -1,
						  2, 0, -2,
							1, 0, -1 };
__managed__ char GY[9] = { 1, 2, 1,
				   0, 0, 0,
				  -1,-2,-1 };


unsigned char* greyScaleBuffer;
int *sobel; //1280x720
unsigned char *finished; //1280x720*3


__global__
void greyScale(unsigned char * frame, unsigned char*greyBuffer) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= WIDTH || y >= HEIGHT)
		return;

	int index = x + y * WIDTH;
	int sum = (frame[3*index] + frame[3*index + 1] + frame[3*index + 2]) / 3;
	greyBuffer[index] = sum;
}


__device__
int index(int x, int y) {
	if (x >= WIDTH || y >= HEIGHT || x < 0 || y < 0)
		return -1;

	return x + y * WIDTH;
}

__device__ __forceinline__
int wrap(int val, int limit) {
	if (val < 0)
		return limit - 1;
	return val % limit;
}

__global__
void sobelOp(unsigned char * greyBuffer, int * sobelBuffer) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= WIDTH || y >= HEIGHT)
		return;

	int xDir = 0;
	int yDir = 0;

	for (int r = -1; r < 2; r++) {
		int row = y + r;
		row = wrap(row, HEIGHT);

		for (int c = -1; c < 2; c++) {
			int col = x + c;
			col = wrap(col, WIDTH);

			xDir += greyBuffer[index(col, row)] * GX[(1 - c) + (1 - r) * 3];
			yDir += greyBuffer[index(col, row)] * GY[(1 - c) + (1 - r) * 3];
		}
	}

	__syncthreads();

	sobelBuffer[2*index(x, y)] = xDir;
	sobelBuffer[2*index(x, y)+1] = yDir;

}

__global__
void render(int* sobolBuffer, unsigned char* frame, unsigned short time) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= WIDTH || y >= HEIGHT)
		return;

	int index = x + y * WIDTH;

	float red = frame[3 * index] / 256.0;
	float green = frame[(3 * index) + 1] / 256.0;
	float blue = frame[(3 * index) + 2] / 256.0;

	int xv = sobolBuffer[2 * index];
	int yv = sobolBuffer[2 * index + 1];
	int mag = (int)sqrt((double) xv * xv + yv * yv);
	if (mag > 255)
		mag = 255;

	frame[3 * index] = red * mag;
	frame[(3 * index) + 1] = green * mag;
	frame[(3 * index) + 2] = blue * mag;
}

void sobelFilter(unsigned char* frame) {
	static unsigned short count = 0;
	dim3 thread(32, 32);
	dim3 block(WIDTH/32 + 1, HEIGHT/32 + 1);
	//dim3 block(40, 23);

	hipMemcpy(finished, frame, SIZE, hipMemcpyHostToDevice);
	greyScale<<<block, thread>>>(finished, greyScaleBuffer); //Convert to grayscale
	sobelOp<<<block, thread>>>(greyScaleBuffer, sobel); //Compute Sobel convolution
	render << <block, thread >> > (sobel, finished, count++);
	count &= 0x3FF;

	hipDeviceSynchronize();
	hipMemcpy(frame, finished, SIZE, hipMemcpyDeviceToHost);
}

void sobelInit() {
	hipMalloc(&greyScaleBuffer, WIDTH * HEIGHT);
	hipMalloc(&sobel, sizeof(int) * WIDTH * HEIGHT * 2);
	hipMalloc(&finished, SIZE);
}

void sobelFree() {
	hipFree(greyScaleBuffer);
	hipFree(sobel);
	hipFree(finished);
}