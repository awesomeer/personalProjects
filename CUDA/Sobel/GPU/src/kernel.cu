#include "hip/hip_runtime.h"
#include "../inc/kernel.h"

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include <stdio.h>


/*
* This code is configured for a GTX 1060M with 1024 CUDA cores
*/


#define HEIGHT 1080
#define WIDTH 1920
#define SIZE (3 * WIDTH * HEIGHT)


unsigned char *gaussian;
unsigned char *finished;


__device__ __forceinline__
int index(int x, int y) {
	if (x >= WIDTH || y >= HEIGHT || x < 0 || y < 0)
		return -1;

	return x + y * WIDTH;
}

__device__ __forceinline__
int wrap(int val, int limit) {
	if (val < 0)
		return limit + val;
	return val % limit;
}



__global__
void greyScale(unsigned char * frame, unsigned char*greyBuffer) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= WIDTH || y >= HEIGHT)
		return;

	int pindex = 3*index(x,y);

	int sum = (0.2126f * ((float)frame[pindex])) 
			+ (0.7152f * ((float)frame[pindex+1]))
			+ (0.0722f * ((float)frame[pindex+2]));

	greyBuffer[pindex] = sum;
	greyBuffer[pindex+1] = sum;
	greyBuffer[pindex+2] = sum;
}


__global__
void sobelOp(unsigned char * frame, unsigned char * sobel) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= WIDTH || y >= HEIGHT)
		return;

	char GX[9] = { 1, 0, -1,
			2, 0, -2,
			1, 0, -1 };
	char GY[9] = { 1, 2, 1,
			   0, 0, 0,
			 -1,-2,-1 };
			 
	for(int p = 0; p < 3; p++){
		int xDir = 0;
		int yDir = 0;

		for (int r = -1; r < 2; r++) {
			int row = y + r;
			row = wrap(row, HEIGHT);

			for (int c = -1; c < 2; c++) {
				int col = x + c;
				col = wrap(col, WIDTH);

				int pindex = 3*index(col, row);

				xDir += frame[pindex+p] * GX[(1 - c) + (1 - r) * 3];
				yDir += frame[pindex+p] * GY[(1 - c) + (1 - r) * 3];
			}
		}

		int pindex = 3*index(x, y);
		int mag = (int)sqrt((double) xDir * xDir + yDir * yDir);
		mag = max(0, min(255, mag));

		sobel[pindex + p] = mag;
	}

}

__global__ void gaussian_filter(const unsigned char *gaussian_input, unsigned char *gaussian_output) {

    const unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;

	if (col >= WIDTH || row >= HEIGHT)
		return;

	 char gaussian_kernel[25] = { 
		1, 4, 6, 4, 1,
		4, 16, 24, 16, 4,
		6, 24, 36, 24, 6,
		4, 16, 24, 16, 4,
		1, 4, 6, 4, 1
	};

	for(int p = 0; p < 3; p++){
		int blur = 0;

		for(int i = -2; i < 3; i++) {
			for(int j = -2; j < 3; j++) {

				const unsigned int y = wrap(row+i, HEIGHT);
				const unsigned int x = wrap(col+j, WIDTH);

				char w = gaussian_kernel[(2-j) + (2-i) * 5];
				int pindex = 3*index(x,y);
				blur += w * gaussian_input[pindex+p];
			}
		}

		blur = min(255, blur/256);
		int pindex = 3*index(col,row);
		gaussian_output[pindex+p] = blur;
	}

}


void filter(unsigned char* frame, FilterType filtertype) {
	dim3 thread(32, 32);
	dim3 block((WIDTH+31)/32, (HEIGHT+31)/32);

	hipMemcpy(gaussian, frame, SIZE, hipMemcpyHostToDevice);

	switch(filtertype){
		case GREY:{
			greyScale<<<block, thread>>>(gaussian, finished);
			break;
		}
		case SOBEL:{
			sobelOp<<<block, thread>>>(gaussian, finished);
			break;
		}
		case GAUSSIAN:{
			for(int i = 0; i < 9; i++){
				gaussian_filter<<<block, thread>>>(gaussian, finished);
				hipMemcpy(gaussian, finished, SIZE, hipMemcpyDeviceToDevice);
			}
			gaussian_filter<<<block, thread>>>(gaussian, finished);
			break;
		}
	}

	hipMemcpy(frame, finished, SIZE, hipMemcpyDeviceToHost);
}


#include <stdio.h>
void initCuda() {
	hipMalloc(&gaussian, SIZE);
	hipMalloc(&finished, SIZE);
}

void freeCuda() {
	hipFree(gaussian);
	hipFree(finished);
}
