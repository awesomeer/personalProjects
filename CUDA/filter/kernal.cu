#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


#define R(row, col, width, height) ((row*width)+col)
#define G(row, col, width, height) ((width*height) + ((row*width)+col))
#define B(row, col, width, height) ((2*width*height) + ((row*width)+col))

#define LDTOLERNACE 1

static unsigned char * dev_dest;
static unsigned char * dev_src;


__global__
void lineDetection(unsigned char * dest, unsigned char * src, int row, int col){
    
    int rowThread = threadIdx.y + blockDim.y * blockIdx.y;
    int colThread = threadIdx.x + blockDim.x * blockIdx.x;
    if(rowThread < 0 || rowThread >= row){
        return;
    }
    else if(colThread < 0 || colThread >= col){
        return;
    }

    int cRed = src[R(rowThread, colThread, col, row)];
    int cGreen = src[G(rowThread, colThread, col, row)];
    int cBlue = src[B(rowThread, colThread, col, row)];

    //right scan
    

    if(colThread < col-1){
        int red = src[R(rowThread, colThread+1, col, row)];
        int green = src[G(rowThread, colThread+1, col, row)];
        int blue = src[B(rowThread, colThread+1, col, row)];
        int average = (abs(cRed-red)+abs(cBlue-blue)+abs(cGreen-green))/3;
        if(average <= LDTOLERNACE){
            dest[R(rowThread, colThread, col, row)] = 0;
            dest[G(rowThread, colThread, col, row)] = 0;
            dest[B(rowThread, colThread, col, row)] = 0;
        }
    }

    if(rowThread < row-1){
        int red = src[R(rowThread+1, colThread, col, row)];
        int green = src[G(rowThread+1, colThread, col, row)];
        int blue = src[B(rowThread+1, colThread, col, row)];
        int average = (abs(cRed-red)+abs(cBlue-blue)+abs(cGreen-green))/3;
        if(average <= LDTOLERNACE){
            dest[R(rowThread, colThread, col, row)] = 0;
            dest[G(rowThread, colThread, col, row)] = 0;
            dest[B(rowThread, colThread, col, row)] = 0;
        }
    }

}





extern void hipLaunchByPtr(unsigned char * dest, unsigned char * src, int row, int col, int filter){

    printf("Line Detection");
    hipMemcpy(dev_src, src, sizeof(unsigned char) * row * col, hipMemcpyHostToDevice);
    dim3 block(32,32);
    dim3 grid((col/32) + 1, (row/32) + 1);
    switch(filter){
       case 0:{
            lineDetection<<<grid, block>>>(dev_dest, dev_src, row, col);
       } 
    }

    hipMemcpy(dest, dev_dest, sizeof(unsigned char) * row * col, hipMemcpyDeviceToHost);

}

extern void initCuda(unsigned long size){
    hipMalloc(&dev_dest, size);
    hipMalloc(&dev_src, size);
}

extern void cudaReset(){
    hipFree(dev_dest);
    hipFree(dev_src);
}