#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

//const int  KEY[] =  {0x32,0x28,0x99};

__global__
void kernal(unsigned char * dest, unsigned char * src, int size){
    const int  KEY[] =  {0x32,0x28,0x99};
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if(index >= size)
        return;
    
    dest[index] = src[index] ^ KEY[index%3];


}

extern void XOR(unsigned char * dest, unsigned char * src, int row, int col){
    unsigned char * dev_src;
    unsigned char * dev_dest;
    int size = sizeof(unsigned char) * 3 * row * col;
    hipMalloc(&dev_src, size);
    hipMalloc(&dev_dest, size);

    hipMemcpy(dev_src, src, size, hipMemcpyHostToDevice);
    printf("nuawdnanowd");
    kernal<<<(size/1024)+1, 1024>>>(dev_dest, dev_src, size);
    hipMemcpy(dest, dev_dest, size, hipMemcpyDeviceToHost);

    hipFree(dev_src);
    hipFree(dev_dest);

}