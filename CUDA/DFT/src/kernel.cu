#include "hip/hip_runtime.h"
#include "..\inc\kernel.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <cmath>
#include <stdio.h>
double * cudaSamples;
double * cudaDFT;
__managed__ int N;

__global__
void kernelDFT(double * samples, double * DFT){
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if(index > N){
        printf("Thread exit\n");
        return;
    }

    double real = 0;
    double imag = 0;
    for(int i = 0; i < N; i++){
        real += samples[i]*cos(2.0*3.14159265* (float) index * (float) i / (float) N);
        imag -= samples[i]*sin(2.0*3.14159265* (float) index * (float) i / (float) N);
    }

    DFT[index] = sqrt(real*real + imag*imag)/(index == 0 ? N : N>>1);
}

void DFT(double * samples){
    hipMemcpy(cudaSamples, samples, sizeof(double)*N, hipMemcpyHostToDevice);
    kernelDFT<<<1, 1024>>>(cudaSamples, cudaDFT);
    hipDeviceSynchronize();
    hipError_t err = hipMemcpy(samples, cudaDFT, sizeof(double)*N, hipMemcpyDeviceToHost);
    if(err)
        printf("%s\n", hipGetErrorString(err));
}

void cudaInit(int num){
    N = num;
    hipMalloc(&cudaSamples, sizeof(double)*N);
    hipMalloc(&cudaDFT, sizeof(double)*N);
}

void cudaExit(){
    hipFree(cudaSamples);
    hipFree(cudaDFT);
}