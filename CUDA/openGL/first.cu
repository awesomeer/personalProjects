#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>





int main(){
	hipDeviceProp_t device;
	hipGetDeviceProperties(&device, 0);
	printf("%s\n", device.name);
	printf("Number of multiprocessors: %d\n", device.multiProcessorCount);
	printf("Global memory: %f GB\n", (double) device.totalGlobalMem/(1024*1024*1024));
	printf("Maximum number threads per block: %d\n", device.maxThreadsPerBlock);
	printf("Maximum number of threads per diminsion x: %d y: %d z: %d\n", device.maxThreadsDim[0], device.maxThreadsDim[1], device.maxThreadsDim[2]);
	printf("Maximum grid size x: %d y: %d z: %d\n", device.maxGridSize[0], device.maxGridSize[1], device.maxGridSize[2]);
	printf("Thread Wrap Size: %d\n", device.warpSize);
}
