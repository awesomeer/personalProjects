#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>




int main(){
	hipDeviceProp_t device;
	hipGetDeviceProperties(&device, 0);
	printf("%s\n", device.name);
	printf("Global memory: %f\n", (double) device.clockRate/1000);
	printf("Struct address: %p\n", &device);

	char input = 0;
	scanf("%c", &input);
	printf("0x%X\n", input);
}
